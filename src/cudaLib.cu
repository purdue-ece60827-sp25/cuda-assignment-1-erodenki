#include "hip/hip_runtime.h"

#include "cudaLib.cuh"
#include <fenv.h>

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ 
void saxpy_gpu (float* x, float* y, float scale, int size) {
    // get current thread index, 1D
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float mul;
    
    // check to make sure inside bounds
    if (idx < size) {
        mul = __fmul_rn(x[idx], scale);
        y[idx] = __fadd_rn(y[idx], mul);
        //y[idx] = x[idx] * scale + y[idx];
    }

    return;
}

int runGpuSaxpy(int vectorSize) {

	std::cout << "Hello GPU Saxpy!\n";

	hipDeviceProp_t devprop;

    // set rounding mode
    fesetround(FE_TONEAREST);
	
	//  Check CUDA device presence
	int numDev;
	hipGetDeviceCount(&numDev);
	if (numDev < 1) {
		std::cout << "CUDA device missing!\n";
		return -1;
    }

	hipGetDeviceProperties(&devprop, 0);

    #ifndef DEBUG_PRINT_DISABLE 
    	printf("Device name: %s\n", devprop.name);
    	printf("Threads per warp: %d\n", devprop.warpSize);
    	printf("Max threads per block: %d\n", devprop.maxThreadsPerBlock);
    	printf("Max thread dims: %d %d %d\n", devprop.maxThreadsDim[0], devprop.maxThreadsDim[1], devprop.maxThreadsDim[2]);
    	printf("Max grid size: %d %d %d\n", devprop.maxGridSize[0], devprop.maxGridSize[1], devprop.maxGridSize[2]);
    #endif

    float *x, *y, *y_new;
    volatile float *scale_mem;
    float scale;

    // allocate space for the arrays
    x = (float *) malloc(vectorSize * sizeof(*x));
    y = (float *) malloc(vectorSize * sizeof(*y));
    y_new = (float *) malloc(vectorSize * sizeof(*y_new));
    scale_mem = (float *) malloc(sizeof(*scale_mem));

    if (x == NULL || y == NULL || y_new == NULL) {
        printf("Memory allocation failed, exiting\n");
        return -1;
    }
    
    srand(time(0));

    // generate a random float for the scale
    //scale = (float) (rand() % 100);
    //scale = (float) (rand()) / (float) (rand());
    *scale_mem = 2.1f;
    scale = *scale_mem;

    // generate random floats for the vectors
    for (int i = 0; i < vectorSize; i++) {
        // might need to change this random gen later to get a wider range
        x[i] = (float) (rand() % 100);
        y[i] = (float) (rand() % 100); 
        y_new[i] = y[i];
    }

	#ifndef DEBUG_PRINT_DISABLE 
        printf("\nScale: %3.4f\n", scale);
        printf("Vectors:\n");
        printf("x: { ");
        for (int i = 0; i < (vectorSize > 10 ? 10 : vectorSize); i++) {
           printf("%3.4f ", x[i]);
        }
        printf("...} \ny: {"); 
        for (int i = 0; i < (vectorSize > 10 ? 10 : vectorSize); i++) {
           printf("%3.4f ", y[i]);
        }
        printf("...} \n");
    #endif

    // generate block num and thread block size
    // use max size thread blocks for now
    int num_threads_per_block = devprop.maxThreadsPerBlock;
    int num_blocks = (vectorSize / num_threads_per_block) + 1;

    // allocate memory and transfer
    float *x_device, *y_device;
    gpuErrchk(hipMalloc(&x_device, vectorSize*sizeof(float)));
    gpuErrchk(hipMalloc(&y_device, vectorSize*sizeof(float)));

    gpuErrchk(hipMemcpy(x_device, x, vectorSize*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(y_device, y_new, vectorSize*sizeof(float), hipMemcpyHostToDevice));

    // invoke kernel
    saxpy_gpu<<<num_blocks, num_threads_per_block>>>(x_device, y_device, scale, vectorSize);

    gpuErrchk(hipMemcpy(y_new, y_device, vectorSize*sizeof(float), hipMemcpyDeviceToHost));

    hipFree(x_device);
    hipFree(y_device);

    int error_count = verifyVector(x, y, y_new, scale, vectorSize);
    printf("%d errors found\n", error_count);

    free(x);
    free(y);
    free(y_new);

    return (0);
}

/* 
 Some helpful definitions

 generateThreadCount is the number of threads spawned initially. Each thread is responsible for sampleSize points. 
 *pSums is a pointer to an array that holds the number of 'hit' points for each thread. The length of this array is pSumSize.

 reduceThreadCount is the number of threads used to reduce the partial sums.
 *totals is a pointer to an array that holds reduced values.
 reduceSize is the number of partial sums that each reduceThreadCount reduces.

*/

__global__
void generatePoints (uint64_t * pSums, uint64_t pSumSize, uint64_t sampleSize) {
	//	Insert code here
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandState_t rng;
    hiprand_init(clock64(), idx, 0, &rng);

    if (idx < pSumSize) {
        for (int i = 0; i < sampleSize; i++) {
            // need to make sure it generates inside the bounds
            float rand_x = hiprand_uniform(&rng);
            float rand_y = hiprand_uniform(&rng);

            if (int(rand_x * rand_x + rand_y * rand_y) == 0) {
                pSums[idx]++;
            }
        }
    }
}

__global__ 
void reduceCounts (uint64_t * pSums, uint64_t * totals, uint64_t pSumSize, uint64_t reduceSize) {
	//	Insert code here
}

int runGpuMCPi (uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {

	//  Check CUDA device presence
	int numDev;
	hipGetDeviceCount(&numDev);
	if (numDev < 1) {
		std::cout << "CUDA device missing!\n";
		return -1;
	}

	auto tStart = std::chrono::high_resolution_clock::now();
		
	float approxPi = estimatePi(generateThreadCount, sampleSize, 
		reduceThreadCount, reduceSize);
	
	std::cout << "Estimated Pi = " << approxPi << "\n";

	auto tEnd= std::chrono::high_resolution_clock::now();

	std::chrono::duration<double> time_span = (tEnd- tStart);
	std::cout << "It took " << time_span.count() << " seconds.";

	return 0;
}

double estimatePi(uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {
	
	double approxPi = 0;
    
    uint64_t *pSums, *d_pSums;

    pSums = (uint64_t *) malloc(generateThreadCount * sizeof(*pSums));

    if (pSums == NULL) {
        printf("Memory allocation failed, exiting\n");
        return -1;    
    }

    hipMalloc(&d_pSums, generateThreadCount*sizeof(*d_pSums));

    int num_threads_per_block = 1024;
    int num_blocks = generateThreadCount / num_threads_per_block + 1;

    generatePoints<<<num_blocks, num_threads_per_block>>>(d_pSums, generateThreadCount, sampleSize);

    hipMemcpy(pSums, d_pSums, generateThreadCount*sizeof(*pSums), hipMemcpyDeviceToHost);
   
    uint64_t sums = 0;

    for (int i = 0; i < generateThreadCount; i++) {
        sums += pSums[i];
    } 

    approxPi = (double) sums / (double) (generateThreadCount * sampleSize) * (double) 4;

    free(pSums);
    hipFree(d_pSums);

	return approxPi;
}
